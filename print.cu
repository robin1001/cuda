
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void print() {
	printf("block = %d, thread = %d\n", blockIdx.x, threadIdx.x);
}

int main() {
	print<<<3,3>>>();
	hipDeviceSynchronize();

}
