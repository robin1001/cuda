
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100

__global__ void assign(int *arr) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;	
	if (tid < N && tid > 0) {
        for (int i = 0; i < 50; i++) {
		    int tmp = arr[tid-1];
		    __syncthreads();
		    arr[tid] = tmp;
		    //arr[tid] = arr[tid-1]; //false operation
        }
	}
}


int main() {
	int arr[N] = {0};
	for (int i = 0; i < N; i++)
		arr[i] = i;
	int b[N] = {0};
	int *dev_arr;
	hipMalloc(&dev_arr, N * sizeof(int));
	hipMemcpy(dev_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);

	assign<<<16, 16>>>(dev_arr);
	hipMemcpy(b, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_arr);

	for (int i = 0; i < N; i++) {
		printf("%d ", b[i]);
	}
	printf("\n");
}
