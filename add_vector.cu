
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512

__global__ void add(int *a, int *b, int *c) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < N) 
		c[tid] = a[tid] + b[tid];
}

int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = 1;
	}
	hipMalloc((void **)&dev_a, N * sizeof(int));
	hipMalloc((void **)&dev_b, N * sizeof(int));
	hipMalloc((void **)&dev_c, N * sizeof(int));
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	int block_dim = 128;
	int grid_dim = N / block_dim;
	add<<<grid_dim, block_dim>>>(dev_a, dev_b, dev_c);
	hipMemcpy(&c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
    return 0;
}
