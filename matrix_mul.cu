
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 64

__global__ void Mul(int *a, int *b, int row_a, int col_a, int row_b, int col_b, int *c) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < row_a * col_b) {
        int value = 0;
        int row  = tid / col_b, col = tid % col_b;
		for (int k = 0; k < col_a; k++)
			value += a[row * col_a + k] * b[k * col_b + col];
		c[tid] = value;
		//c[tid] = 1;
	}
}

int main() {
    int data_a[3][3] = {0}, data_b[3][3] = {0}, data_c[3][3];
    int *dev_a, *dev_b, *dev_c;
    data_a[0][0] = 1, data_a[1][1] = 1, data_a[2][2] = 1;
    data_b[0][0] = 2, data_b[1][1] = 1, data_b[2][2] = 1;

	hipMalloc((void **)&dev_a, 3 * 3 * sizeof(int));
	hipMemcpy(dev_a, data_a, 3 * 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_b, 3 * 3 * sizeof(int));
	hipMemcpy(dev_b, data_b, 3 * 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&dev_c, 3 * 3 * sizeof(int));
    Mul<<<10, 10>>>(dev_a, dev_b, 3, 3, 3, 3, dev_c);
	hipMemcpy(data_c, dev_c, 3 * 3 * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
	
    for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			printf("%d ", data_c[i][j]);
		}
		printf("\n");
	}
}


