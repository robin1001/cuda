
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 16

__global__ void assign(int *arr, int *r) {
	__shared__ int data[N];
	int tid = threadIdx.x;	
	if (tid < N) {
		data[tid] = arr[tid];
		__syncthreads();
		for (int i = blockDim.x / 2; i != 0; i /= 2) {
			if (tid < i) {
				data[tid] += data[tid+i];	
				__syncthreads();
			}
		}
	}
	if (blockIdx.x == 0)	
		*r = data[0];
}


int main() {
	int arr[N] = {0};
	int *dev_arr;
	int *dev_r;
	int r, v = 0;
	for (int i = 0; i < N; i++)	{
		arr[i] = i;
		v += arr[i];
	}
	hipMalloc(&dev_arr, N * sizeof(int));
	hipMemcpy(dev_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&dev_r, sizeof(int));

	assign<<<1, N>>>(dev_arr, dev_r);
	hipMemcpy(&r, dev_r, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_arr);
	hipFree(dev_r);

	printf("%d %d\n", r, v);
	assert(r == v);
}
